#include "hip/hip_runtime.h"
#include "ssm_scan.cuh"

template <int block_size>
static __global__ void ssm_scan_f32(
    const float * src0, const float * src1, const float * src2, const float * src3,
    const float * src4, const float * src5,
    const int src0_nb1, const int src0_nb2,
    const int src1_nb0, const int src1_nb1, const int src1_nb2, const int src1_nb3,
    const int src2_nb0, const int src2_nb1, const int src2_nb2,
    const int src3_nb1,
    const int src4_nb1, const int src4_nb2,
    const int src5_nb1, const int src5_nb2,
    float * dst,
    const int nc, const int nr) {

    const int tid = threadIdx.x;
    const int i2 = blockIdx.x;
    const int i3 = threadIdx.y;

    const int ith = tid;
    const int nth = WARP_SIZE;

    // rows per thread
    const int dr = (nr + nth - 1)/nth;

    // row range for this thread
    const int ir0 = dr*ith;
    const int ir1 = min(ir0 + dr, nr);
    const int ir  = ir1 - ir0;

    const float * s0 = (const float *) ((const char *) src0 + ir0*src0_nb1 + i3*src0_nb2); // {d_state, d_inner, n_s}
    const float * x  = (const float *) ((const char *) src1 + ir0*src1_nb0 + i2*src1_nb1 + i3*src1_nb2); // {d_inner, n_t, n_s}
    const float * dt = (const float *) ((const char *) src2 + ir0*src2_nb0 + i2*src2_nb1 + i3*src2_nb2); // {d_inner, n_t, n_s}
    const float * A  = (const float *) ((const char *) src3 + ir0*src3_nb1); // {d_state, d_inner}
    const float * B  = (const float *) ((const char *) src4 +  i2*src4_nb1 + i3*src4_nb2); // {d_state, n_t, n_s}
    const float * C  = (const float *) ((const char *) src5 +  i2*src5_nb1 + i3*src5_nb2); // {d_state, n_t, n_s}
    float * y = (float *) ((char *) dst + ir0*src1_nb0 + i2*src1_nb1 + i3*src1_nb2); // {d_inner, n_t, n_s}
    float * s = (float *) ((char *) dst + ir0*src0_nb1 + i3*src0_nb2 + src1_nb3); // {d_state, d_inner, n_s}

    // use the output as the source for the next token-wise iterations
    if (i2 > 0) { s0 = s; }

    // d_inner
    #pragma unroll
    for (int i1 = 0; i1 < ir; ++i1) {
        // ref: https://github.com/state-spaces/mamba/blob/34076d664838588a3c97727b263478ab9f621a07/mamba_ssm/ops/triton/selective_state_update.py#L78
        float dt_soft_plus = dt[i1] <= 20.0f ? log1pf(expf(dt[i1])) : dt[i1];
        float x_dt = x[i1] * dt_soft_plus;
        float sumf = 0.0f;
        // d_state
        #pragma unroll
        for (int i0 = 0; i0 < nc; ++i0) {
            int i = i0 + i1*nc;
            // state = prev_state * dA + dB * x
            float state = (s0[i] * expf(dt_soft_plus * A[i])) + (B[i0] * x_dt);
            // y = rowwise_dotprod(state, C)
            sumf += state * C[i0];
            s[i] = state;
        }
        y[i1] = sumf;
    }
}

static void ssm_scan_f32_cuda(
    const float * src0, const float * src1, const float * src2, const float * src3,
    const float * src4, const float * src5,
    const int src0_nb1, const int src0_nb2,
    const int src1_nb0, const int src1_nb1, const int src1_nb2, const int src1_nb3,
    const int src2_nb0, const int src2_nb1, const int src2_nb2,
    const int src3_nb1,
    const int src4_nb1, const int src4_nb2,
    const int src5_nb1, const int src5_nb2,
    float * dst,
    const int nc, const int nr, const int n_t, const int n_s,
    hipStream_t stream) {

    const dim3 block_dims(WARP_SIZE, n_s, 1);
    const int nblocks = n_t;

    ssm_scan_f32<WARP_SIZE><<<nblocks, block_dims, 0, stream>>>(
        src0, src1, src2, src3,
        src4, src5,
        src0_nb1, src0_nb2,
        src1_nb0, src1_nb1, src1_nb2, src1_nb3,
        src2_nb0, src2_nb1, src2_nb2,
        src3_nb1,
        src4_nb1, src4_nb2,
        src5_nb1, src5_nb2,
        dst,
        nc, nr);
}

void ggml_cuda_op_ssm_scan(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const struct ggml_tensor * src0 = dst->src[0]; // s
    const struct ggml_tensor * src1 = dst->src[1]; // x
    const struct ggml_tensor * src2 = dst->src[2]; // dt
    const struct ggml_tensor * src3 = dst->src[3]; // A
    const struct ggml_tensor * src4 = dst->src[4]; // B
    const struct ggml_tensor * src5 = dst->src[5]; // C

    const int64_t nc  = src0->ne[0]; // d_state
    const int64_t nr  = src0->ne[1]; // d_inner
    const int64_t n_t = src1->ne[1]; // number of tokens per sequence
    const int64_t n_s = src0->ne[2]; // number of sequences in the batch

    GGML_ASSERT(ggml_nelements(src1) + ggml_nelements(src0) == ggml_nelements(dst));
    GGML_ASSERT(src0->nb[0] == sizeof(float));
    GGML_ASSERT(src1->nb[0] == sizeof(float));
    GGML_ASSERT(src2->nb[0] == sizeof(float));
    GGML_ASSERT(src3->nb[0] == sizeof(float));
    GGML_ASSERT(src4->nb[0] == sizeof(float));
    GGML_ASSERT(src5->nb[0] == sizeof(float));
    // required for the dot product between s and C
    GGML_ASSERT(src0->nb[1] == src0->ne[0]*sizeof(float));
    // required for per-sequence offsets for states
    GGML_ASSERT(src0->nb[2] == src0->ne[0]*src0->ne[1]*sizeof(float));
    // required to get correct offset for state destination (i.e. src1->nb[3])
    GGML_ASSERT(src1->nb[3] == src1->ne[0]*src1->ne[1]*src1->ne[2]*sizeof(float));

    const float * src0_d = (const float *)src0->data;
    const float * src1_d = (const float *)src1->data;
    const float * src2_d = (const float *)src2->data;
    const float * src3_d = (const float *)src3->data;
    const float * src4_d = (const float *)src4->data;
    const float * src5_d = (const float *)src5->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    ssm_scan_f32_cuda(
        src0_d, src1_d, src2_d, src3_d,
        src4_d, src5_d,
        src0->nb[1], src0->nb[2],
        src1->nb[0], src1->nb[1], src1->nb[2], src1->nb[3],
        src2->nb[0], src2->nb[1], src2->nb[2],
        src3->nb[1],
        src4->nb[1], src4->nb[2],
        src5->nb[1], src5->nb[2],
        dst_d,
        nc, nr, n_t, n_s,
        stream);
}
